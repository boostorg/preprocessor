# /* **************************************************************************
#  *                                                                          *
#  *     (C) Copyright Edward Diener 2019.
#  *     Distributed under the Boost Software License, Version 1.0. (See
#  *     accompanying file LICENSE_1_0.txt or copy at
#  *     http://www.boost.org/LICENSE_1_0.txt)
#  *                                                                          *
#  ************************************************************************** */
#
# /* See http://www.boost.org for most recent version. */
#

#if defined(__clang__) && defined(__HIPCC__) && defined(__CUDA__)

static_assert(BOOST_PP_VARIADICS == 1, "");

#else

static_assert(0 == 1, "");

#endif

int main(void) {
    return 0;
}
